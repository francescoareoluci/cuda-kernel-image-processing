#include <iostream>
#include <chrono>
#include "image.h"
#include "kernel.h"

int main(int argc, char **argv)
{
	printf("===== Multithread kernel convolution =====\n");

	// The first call to the CUDA device will take a lot of time,
	// better do it here
	hipFree(0);

	Kernel kernel;
	//kernel.setSharpenFilter();
	//kernel.setGaussianFilter(25, 25, 1);
	//kernel.setEdgeDetectionFilter();
	kernel.setGaussianLaplacianFilter();

	Image img;
	img.loadImage("images/1.png");

	Image newMtImg;
	Image newNpImg;

	// Executing multithread filtering for each image
	auto t1 = std::chrono::high_resolution_clock::now();
	img.multithreadFiltering(newMtImg, kernel);
	auto t2 = std::chrono::high_resolution_clock::now();

	auto t3 = std::chrono::high_resolution_clock::now();
	img.applyFilter(newNpImg, kernel);
	auto t4 = std::chrono::high_resolution_clock::now();

	// Evaluating execution times
	auto multithreadDuration = std::chrono::duration_cast<std::chrono::microseconds>( t2 - t1 ).count();
	auto singleDuration = std::chrono::duration_cast<std::chrono::microseconds>( t4 - t3 ).count();

	std::cout << "Multithread Execution time: " << multithreadDuration << std::endl;
	std::cout << "Single thread Execution time: " << singleDuration << std::endl;

	newMtImg.saveImage("output/1_mt.png");
	newNpImg.saveImage("output/1_np.png");
}
