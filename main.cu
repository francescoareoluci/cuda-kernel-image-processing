#include <iostream>
#include <chrono>
#include "image.h"
#include "kernel.h"

#define GAUSSIAN_FILTER_COMMAND             "gaussian"
#define SHARPENING_FILTER_COMMAND         "sharpen"
#define EDGE_DETECTION_FILTER_COMMAND  "edge_detect"
#define LAPLACIAN_FILTER_COMMAND            "laplacian"
#define GAUSSIAN_LAPLACIAN_COMMAND      "gaussian_laplacian"

#define CUDA_GLOBAL		"global"
#define CUDA_CONSTANT	"constant"
#define CUDA_SHARED		"shared"

enum class FilterType
{
    GAUSSIAN_FILTER,
    SHARPEN_FILTER,
    EDGE_DETECTION,
    LAPLACIAN_FILTER,
    GAUSSIAN_LAPLACIAN_FILTER
};

int main(int argc, char **argv)
{
	std::cout << "===== Multithread kernel convolution =====" << std::endl;

	 // Check command line parameters
	 if (argc < 3) {
		 std::cerr << "Usage: " << argv[0] << " filter_type image_path cuda_mem_tye" << std::endl;
		 std::cerr << "filter_type: <gaussian | sharpen | edge_detect | alt_edge_detect>" << std::endl;
	    std::cerr << "image_path: specify the image path" << std::endl;
	    std::cerr << "(optional) cuda_mem_type: <global | constant | shared>. Default: shared" << std::endl;
	     return 1;
	}

	 FilterType filterType;
	 std::string cmdFilter = std::string(argv[1]);
	  if (cmdFilter == GAUSSIAN_FILTER_COMMAND) {
		  filterType = FilterType::GAUSSIAN_FILTER;
	 }
	 else if (cmdFilter == SHARPENING_FILTER_COMMAND) {
		  filterType = FilterType::SHARPEN_FILTER;
	 }
	 else if (cmdFilter == EDGE_DETECTION_FILTER_COMMAND) {
		 filterType = FilterType::EDGE_DETECTION;
	 }
	 else if (cmdFilter == LAPLACIAN_FILTER_COMMAND) {
		 filterType = FilterType::LAPLACIAN_FILTER;
	 }
	 else if (cmdFilter == GAUSSIAN_LAPLACIAN_COMMAND) {
		 filterType = FilterType::GAUSSIAN_LAPLACIAN_FILTER;
	 }
	 else {
		 std::cerr << "Invalid filter type " << cmdFilter << std::endl;
	    std::cerr << "filter_type: <gaussian | sharpen | edge_detect | laplacian | gaussian_laplacian >" << std::endl;
	     return 1;
	}

   Kernel filter = Kernel();
	switch (filterType)
	{
	 	 case FilterType::GAUSSIAN_FILTER:
	 		 filter.setGaussianFilter(7, 7, 1);
	     break;

	     case FilterType::SHARPEN_FILTER:
	    	 filter.setSharpenFilter();
	     break;

	     case FilterType::EDGE_DETECTION:
	    	 filter.setEdgeDetectionFilter();
	     break;

	     case FilterType::LAPLACIAN_FILTER:
	    	 filter.setLaplacianFilter();
	     break;

	     case FilterType::GAUSSIAN_LAPLACIAN_FILTER:
	    	 filter.setGaussianLaplacianFilter();
	     break;

	     default:
	    	 std::cerr << "Unable to find requested filter, switching to gaussian..." << std::endl;
	       filter.setGaussianFilter(5, 5, 2);
	      break;
	}
	filter.printKernel();

	CudaMemType cudaType = CudaMemType::SHARED;
	if (argc == 4) {
		std::string cudaMemCmd = std::string(argv[3]);
		if (cudaMemCmd == CUDA_GLOBAL)
			cudaType = CudaMemType::GLOBAL;
		else if(cudaMemCmd == CUDA_CONSTANT)
			cudaType = CudaMemType::CONSTANT;
		else if(cudaMemCmd == CUDA_SHARED)
			cudaType = CudaMemType::SHARED;
	}

	Image img;
	bool loadResult = img.loadImage(argv[2]);
	if (!loadResult) {
		std::cerr << "Unable to load image " << argv[2] << std::endl;
		return 1;
	}

	Image newMtImg;
	Image newNpImg;

	// The first call to the CUDA device will take a lot of time,
	// better do it here
	hipFree(0);

	// Executing multithread filtering for each image
	auto t1 = std::chrono::high_resolution_clock::now();
	bool cudaResult = img.multithreadFiltering(newMtImg, filter, cudaType);
	auto t2 = std::chrono::high_resolution_clock::now();

	auto t3 = std::chrono::high_resolution_clock::now();
	bool sequentialResult = img.applyFilter(newNpImg, filter);
	auto t4 = std::chrono::high_resolution_clock::now();

	// Evaluating execution times and save results
	if (cudaResult) {
		auto multithreadDuration = std::chrono::duration_cast<std::chrono::microseconds>( t2 - t1 ).count();
		std::cout << "Multithread Execution time: " << multithreadDuration << " μs" << std::endl;
		newMtImg.saveImage("output/1_mt.png");
	}

	if (sequentialResult) {
		auto singleDuration = std::chrono::duration_cast<std::chrono::microseconds>( t4 - t3 ).count();
		std::cout << "Sequential Execution time: " << singleDuration << " μs" << std::endl;
		newNpImg.saveImage("output/1_np.png");
	}
}
