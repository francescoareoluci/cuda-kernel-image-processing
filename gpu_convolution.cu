#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <chrono>
#include "gpu_convolution.h"
#include "hip/hip_runtime.h"

unsigned int divUp(const unsigned int& a, const unsigned int& b)
{
	if (a % b != 0) {
		return a / b + 1;
	}
	else {
		return a / b;
	}
}

const unsigned int MAX_FILTER_SIZE = 25;
__device__ __constant__ float d_cFilterKernel[MAX_FILTER_SIZE * MAX_FILTER_SIZE];

__global__ void filterImageGlobal(float* d_sourceImagePtr, float* d_maskPtr, float* d_outImagePtr,
						int width, int height, int paddedWidth, int paddedHeight,
						int filterWidth, int filterHeight)
{
	const int s = floor(static_cast<float>(filterWidth) / 2);
	const int i = blockIdx.y * blockDim.y + threadIdx.y + s;
	const int j = blockIdx.x * blockDim.x + threadIdx.x + s;

	unsigned int filterRowIndex = 0;
	unsigned int sourceImgRowIndex = 0;
	unsigned int sourceImgIndex = 0;
	unsigned int maskIndex = 0;
	float pixelSum = 0;

	// Check out of bounds thread idx
	if( j >= s && j < paddedWidth - s &&
			i >= s && i < paddedHeight - s) {

		int outPixelPos = (j - s) + (i - s) * width;

		// Apply convolution
		for (int h = -s;  h <= s; h++) {
			filterRowIndex = (h + s) * filterWidth;
	    	sourceImgRowIndex = (h + i) * paddedWidth;
	    	for (int w = -s; w <= s; w++) {
	    		sourceImgIndex = w + j + sourceImgRowIndex;
	    		maskIndex = (w + s) + filterRowIndex;
	    		pixelSum += d_sourceImagePtr[sourceImgIndex] * d_maskPtr[maskIndex];
	    	}
		}

		// Thresholding overflowing pixel's values
		if (pixelSum < 0) {
			pixelSum = 0;
		}
		else if (pixelSum > 255) {
			pixelSum = 255;
		}

		// Write pixel on the output image
		d_outImagePtr[outPixelPos] = pixelSum;
		pixelSum = 0;
	}
}

__global__ void filterImageConstant(float* d_sourceImagePtr, float* d_outImagePtr,
						int width, int height, int paddedWidth, int paddedHeight,
						int filterWidth, int filterHeight)
{
	const int s = floor(float(filterWidth) / 2);
	const int i = blockIdx.y * blockDim.y + threadIdx.y + s;
	const int j = blockIdx.x * blockDim.x + threadIdx.x + s;

	unsigned int filterRowIndex = 0;
	unsigned int sourceImgRowIndex = 0;
	float pixelSum = 0;
	unsigned int sourceImgIndex = 0;
	unsigned int maskIndex = 0;

	// Check out of bounds thread idx
	if( j >= s && j < paddedWidth - s &&
			i >= s && i < paddedHeight - s) {

		int outPixelPos = (j - s) + (i - s) * width;

		// Apply convolution
		for (int h = -s;  h <= s; h++) {
			filterRowIndex = (h + s) * filterWidth;
	    	sourceImgRowIndex = (h + i) * paddedWidth;
	    	for (int w = -s; w <= s; w++) {
	    		sourceImgIndex = w + j + sourceImgRowIndex;
	    		maskIndex = (w + s) + filterRowIndex;
	    		// Here we use the kernel in constant memory
	    		pixelSum += d_sourceImagePtr[sourceImgIndex] * d_cFilterKernel[maskIndex];
	    	}
		}

		// Thresholding overflowing pixel's values
		if (pixelSum < 0) {
			pixelSum = 0;
		}
		else if (pixelSum > 255) {
			pixelSum = 255;
		}

		// Write pixel on the output image
		d_outImagePtr[outPixelPos] = pixelSum;
		pixelSum = 0;
	}
}

__global__ void filterImageShared(float* d_sourceImagePtr, float* d_outImagePtr,
									int paddedWidth, int paddedHeight,
									int blockWidth, int blockHeight,
									int surroundingPixels,
									int width, int height,
									int filterWidth, int filterHeight)
{
	// Each block will share the same data, enabling a faster memory access.
	// Global memory access for each block will be: number of tile sub blocks * threads
	// instead of 9 * threads

	// Tile shared array (dynamically sized by kernel launcher)
	extern __shared__ float s_data[];

	// Evaluate tile's size
	unsigned int tileWidth = blockWidth + 2 * surroundingPixels;
	unsigned int tileHeight = blockHeight + 2 * surroundingPixels;

	// Evaluates number of sub blocks
	unsigned int noSubBlocks = static_cast<int>(ceil(static_cast<float>(tileHeight) /
																					static_cast<float>(blockDim.y)));

	// Get start and end coordinates for blocks
	unsigned int blockStartCol = blockIdx.x * blockWidth + surroundingPixels;
	unsigned int blockEndCol = blockStartCol + blockWidth;
	unsigned int blockStartRow = blockIdx.y * blockHeight + surroundingPixels;
	unsigned int blockEndRow = blockStartRow + blockHeight;

	// Get start and end coordinates for tiles
	unsigned int tileStartCol = blockStartCol - surroundingPixels;
	unsigned int tileEndCol = blockEndCol + surroundingPixels;
	unsigned int tileEndClampedCol = min(tileEndCol, paddedWidth);

	unsigned int tileStartRow = blockStartRow - surroundingPixels;
	unsigned int tileEndRow = blockEndRow + surroundingPixels;
	unsigned int tileEndClampedRow = min(tileEndRow, paddedHeight);

	// Pixel position in tile
	unsigned int tilePixelPosCol = threadIdx.x;
	// Input image pixel column position
	unsigned int iPixelPosCol = tileStartCol + tilePixelPosCol;

	unsigned int tilePixelPosRow = 0;
	unsigned int iPixelPosRow = 0;
	unsigned int iPixelPos = 0;
	unsigned int tilePixelPos = 0;

	for (int subBlockNo = 0; subBlockNo < noSubBlocks; subBlockNo++) {
		tilePixelPosRow = threadIdx.y + subBlockNo * blockDim.y;
		iPixelPosRow = tileStartRow + tilePixelPosRow;

		// Check if the pixel is in the image
		if (iPixelPosCol < tileEndClampedCol && iPixelPosRow < tileEndClampedRow) {
			iPixelPos = iPixelPosRow * paddedWidth + iPixelPosCol;
	      tilePixelPos = tilePixelPosRow * tileWidth + tilePixelPosCol;
	      // Load the pixel in the shared memory
	      s_data[tilePixelPos] = d_sourceImagePtr[iPixelPos];
	    }
	}

	// Wait for threads loading data in tiles
	__syncthreads();

	if (iPixelPosCol >= tileStartCol + surroundingPixels &&
	    		iPixelPosCol < tileEndClampedCol - surroundingPixels) {

	unsigned int oPixelPosRow = 0;
	unsigned int oPixelPos = 0;;
	unsigned int tilePixelPosOffset = 0;
	unsigned int maskIndex = 0;
	unsigned int oPixelPosCol = iPixelPosCol - surroundingPixels;

	for (int subBlockNo = 0; subBlockNo < noSubBlocks; subBlockNo++) {

		 float pixelSum = 0;
	    tilePixelPosRow = threadIdx.y + subBlockNo * blockDim.y;
	    iPixelPosRow = tileStartRow + tilePixelPosRow;

	    // Check if the pixel is in the tile and image.
	    // Pixels in the tile padding are exclude from evaluation.
	     if (iPixelPosRow >= tileStartRow + surroundingPixels &&
	    		iPixelPosRow < tileEndClampedRow - surroundingPixels) {

	    	 // Evaluate pixel position for output image
	    	 //oPixelPosCol = iPixelPosCol - surroundingPixels;
	    	 oPixelPosRow = iPixelPosRow - surroundingPixels;
	    	 oPixelPos = oPixelPosRow * width + oPixelPosCol;

	    	 tilePixelPos = tilePixelPosRow * tileWidth + tilePixelPosCol;

	    	 // Apply convolution
	    	 for (int h = -surroundingPixels;  h <= surroundingPixels; h++) {
	    		 for (int w = -surroundingPixels; w <= surroundingPixels; w++) {
	    			 tilePixelPosOffset = h * tileWidth + w;
	    			 maskIndex = (h + surroundingPixels) * filterWidth + (w + surroundingPixels);
	    			 pixelSum += s_data[tilePixelPos + tilePixelPosOffset] * d_cFilterKernel[maskIndex];
				}
			}

	    	// Thresholding overflowing pixel's values
			if (pixelSum < 0) {
				pixelSum = 0;
			}
			else if (pixelSum > 255) {
				pixelSum = 255;
			}

			// Write pixel on the output image
			d_outImagePtr[oPixelPos] = pixelSum;
			pixelSum = 0;
	    }
	}
	}
}

bool run(const float* sourceImage,
        float* outImage,
        const float* mask,
        int width, int height,
        int paddedWidth, int paddedHeight,
        int filterWidth, int filterHeight)
{
	std::cout << "Starting CUDA global memory convolution" << std::endl;

	const int blockWidth = 32;
	const int blockHeight = 32;

	float *d_sourceImagePtr;
	float *d_outImagePtr;
	float *d_maskPtr;

	const int sourceImgSize = sizeof(float) * paddedWidth * paddedHeight;
	const int maskSize = sizeof(float) * filterWidth * filterHeight;
	const int outImageSize = sizeof(float) * width * height;

	int copyDuration = 0;
	auto t3 = std::chrono::high_resolution_clock::now();

	// Allocate device memory for images and filter
	hipMalloc(reinterpret_cast<void**>(&d_sourceImagePtr), sourceImgSize);
	hipMalloc(reinterpret_cast<void**>(&d_maskPtr), maskSize);
	hipMalloc(reinterpret_cast<void**>(&d_outImagePtr), outImageSize);

	auto t4 = std::chrono::high_resolution_clock::now();
	copyDuration += std::chrono::duration_cast<std::chrono::microseconds>( t4 - t3 ).count();

	hipError_t err = hipGetLastError();
	if (err != hipSuccess) {
		printf("Error: %s\n", hipGetErrorString(err));
		return false;
	}

	t3 = std::chrono::high_resolution_clock::now();

	// Transfer data from host to device memory
	hipMemcpy(d_sourceImagePtr, sourceImage, sourceImgSize, hipMemcpyHostToDevice);
	hipMemcpy(d_maskPtr, mask, maskSize, hipMemcpyHostToDevice);

	t4 = std::chrono::high_resolution_clock::now();
	copyDuration += std::chrono::duration_cast<std::chrono::microseconds>( t4 - t3 ).count();

	err = hipGetLastError();
	if (err != hipSuccess) {
		printf("Error: %s\n", hipGetErrorString(err));
		return false;
	}

	// Allocates block size and grid size
	dim3 threadsPerBlock(blockWidth, blockHeight);
	dim3 blocksPerGrid(divUp(width, blockWidth), divUp(height, blockHeight));

	printf("Blocks: %d, Threads: %d\n", width / threadsPerBlock.x * height / threadsPerBlock.y, blockWidth * blockHeight);

	auto t1 = std::chrono::high_resolution_clock::now();

	filterImageGlobal<<<blocksPerGrid, threadsPerBlock>>>(d_sourceImagePtr, d_maskPtr, d_outImagePtr,
					 width,  height,  paddedWidth,  paddedHeight,
					 filterWidth,  filterHeight);

	err = hipGetLastError();
	if (err != hipSuccess) {
	    printf("Error: %s\n", hipGetErrorString(err));
	    return false;
	}

	// Waits for threads to finish work
	hipDeviceSynchronize();

	auto t2 = std::chrono::high_resolution_clock::now();
	auto filterDuration = std::chrono::duration_cast<std::chrono::microseconds>( t2 - t1 ).count();
	std::cout << "CUDA global memory filtering execution time: " << filterDuration << " μs" << std::endl;

	t3 = std::chrono::high_resolution_clock::now();

	// Transfer resulting image back
	hipMemcpy(outImage, d_outImagePtr, outImageSize, hipMemcpyDeviceToHost);

	t4 = std::chrono::high_resolution_clock::now();
	copyDuration += std::chrono::duration_cast<std::chrono::microseconds>( t4 - t3 ).count();
	std::cout << "Copy Execution time: " << copyDuration << " μs" << std::endl;

	// Cleanup after kernel execution
	hipFree(d_sourceImagePtr);
	hipFree(d_maskPtr);
	hipFree(d_outImagePtr);

	return true;
}

bool runConstant(const float* sourceImage,
        float* outImage,
        const float* mask,
        int width, int height,
        int paddedWidth, int paddedHeight,
        int filterWidth, int filterHeight)
{
	std::cout << "Starting CUDA constant memory convolution" << std::endl;

	const int blockWidth = 32;
	const int blockHeight = 32;

	float *d_sourceImagePtr;
	float *d_outImagePtr;

	const int sourceImgSize = sizeof(float) * paddedWidth * paddedHeight;
	const int maskSize = sizeof(float) * filterWidth * filterHeight;
	const int outImageSize = sizeof(float) * width * height;

	int copyDuration = 0;
	auto t3 = std::chrono::high_resolution_clock::now();

	// Allocate device memory for images and filter
	hipMalloc(reinterpret_cast<void**>(&d_sourceImagePtr), sourceImgSize);
	hipMalloc(reinterpret_cast<void**>(&d_outImagePtr), outImageSize);

	auto t4 = std::chrono::high_resolution_clock::now();
	copyDuration += std::chrono::duration_cast<std::chrono::microseconds>( t4 - t3 ).count();

	hipError_t err = hipGetLastError();
	if (err != hipSuccess) {
		printf("Error: %s\n", hipGetErrorString(err));
		return false;
	}

	t3 = std::chrono::high_resolution_clock::now();

	// Transfer data from host to device memory
	hipMemcpy(d_sourceImagePtr, sourceImage, sourceImgSize, hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(d_cFilterKernel), mask, maskSize, 0, hipMemcpyHostToDevice);

	t4 = std::chrono::high_resolution_clock::now();
	copyDuration += std::chrono::duration_cast<std::chrono::microseconds>( t4 - t3 ).count();

	err = hipGetLastError();
	if (err != hipSuccess) {
		printf("Error: %s\n", hipGetErrorString(err));
		return false;
	}

	// Allocates block size and grid size
	dim3 threadsPerBlock(blockWidth, blockHeight);
	dim3 blocksPerGrid(divUp(width, blockWidth), divUp(height, blockHeight));

	printf("Blocks: %d, Threads: %d\n", width / threadsPerBlock.x * height / threadsPerBlock.y, blockWidth * blockHeight);

	auto t1 = std::chrono::high_resolution_clock::now();

	filterImageConstant<<<blocksPerGrid, threadsPerBlock>>>(d_sourceImagePtr, d_outImagePtr,
					 width,  height,  paddedWidth,  paddedHeight,
					 filterWidth,  filterHeight);

	err = hipGetLastError();
	if (err != hipSuccess) {
	    printf("Error: %s\n", hipGetErrorString(err));
	    return false;
	}

	// Waits for threads to finish work
	hipDeviceSynchronize();

	auto t2 = std::chrono::high_resolution_clock::now();
	auto filterDuration = std::chrono::duration_cast<std::chrono::microseconds>( t2 - t1 ).count();
	std::cout << "CUDA constant filtering execution time: " << filterDuration << " μs" << std::endl;

	// Transfer resulting image back
	t3 = std::chrono::high_resolution_clock::now();
	hipMemcpy(outImage, d_outImagePtr, outImageSize, hipMemcpyDeviceToHost);
	t4 = std::chrono::high_resolution_clock::now();
	copyDuration += std::chrono::duration_cast<std::chrono::microseconds>( t4 - t3 ).count();
	std::cout << "Copy Execution time: " << copyDuration << " μs" << std::endl;

	// Cleanup after kernel execution
	hipFree(d_sourceImagePtr);
	hipFree(d_outImagePtr);

	return true;
}

bool runShared(const float* sourceImage,
        		float* outImage,
        		const float* mask,
        		int width, int height,
        		int paddedWidth, int paddedHeight,
        		int filterWidth, int filterHeight)
{
	std::cout << "Starting CUDA shared memory convolution" << std::endl;

	float *d_sourceImagePtr;
	float *d_outImagePtr;

	const int blockWidth = 64;
	const int blockHeight = 32;
	const int surroundingPixels = floor(filterWidth / 2);

	// TIles includes block size + block padding
	const int tileWidth = blockWidth + 2 * surroundingPixels;
	const int tileHeight = blockHeight + 2 * surroundingPixels;

	// Thread block height will be less than its width.
	// This way we can use bigger kernel size without
	// exceeding thread limit
	const int threadBlockHeight = 4;

	// Evaluate images and kernel size
	const int sourceImgSize = sizeof(float) * paddedWidth * paddedHeight;
	const int maskSize = sizeof(float) * filterWidth * filterHeight;
	const int outImageSize = sizeof(float) * width * height;

	dim3 threadsPerBlock(tileWidth, threadBlockHeight);
	dim3 blocksPerGrid(divUp(width, blockWidth),
											divUp(height, blockHeight));

	int noSubBlocks = static_cast<int>(ceil(static_cast<float>(tileHeight) /
																					static_cast<float>(divUp(height, blockHeight))));

	printf("Blocks: %d, Threads: %d\n", divUp(width, blockWidth) * divUp(height, blockHeight), tileWidth * threadBlockHeight);

	// Evaluates the shared memory size
	int sharedMemorySize = tileWidth * tileHeight * sizeof(float);

	int copyDuration = 0;
	auto t3 = std::chrono::high_resolution_clock::now();

	// Allocate device memory for images
	hipMalloc(reinterpret_cast<void**>(&d_sourceImagePtr), sourceImgSize);
	hipMalloc(reinterpret_cast<void**>(&d_outImagePtr), outImageSize);

	auto t4 = std::chrono::high_resolution_clock::now();
	copyDuration += std::chrono::duration_cast<std::chrono::microseconds>( t4 - t3 ).count();

	hipError_t err = hipGetLastError();
	if (err != hipSuccess) {
		printf("Error: %s\n", hipGetErrorString(err));
		return false;
	}

	t3 = std::chrono::high_resolution_clock::now();

	// Transfer data from host to device memory
	hipMemcpy(d_sourceImagePtr, sourceImage, sourceImgSize, hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(d_cFilterKernel), mask, maskSize, 0, hipMemcpyHostToDevice);

	t4 = std::chrono::high_resolution_clock::now();
	copyDuration += std::chrono::duration_cast<std::chrono::microseconds>( t4 - t3 ).count();

	err = hipGetLastError();
	if (err != hipSuccess) {
		printf("Error: %s\n", hipGetErrorString(err));
		return false;
	}

	auto t1 = std::chrono::high_resolution_clock::now();

	// Launch kernel specifying the shared memory size
	filterImageShared<<<blocksPerGrid, threadsPerBlock, sharedMemorySize>>>(d_sourceImagePtr, d_outImagePtr,
																				paddedWidth, paddedHeight,
																				blockWidth, blockHeight,
																				surroundingPixels,
																				width, height,
																				filterWidth, filterHeight);

	err = hipGetLastError();
	if (err != hipSuccess) {
		printf("Error: %s\n", hipGetErrorString(err));
		return false;
	}

	// Waits for threads to finish work
	hipDeviceSynchronize();

	auto t2 = std::chrono::high_resolution_clock::now();
	auto filterDuration = std::chrono::duration_cast<std::chrono::microseconds>( t2 - t1 ).count();
	std::cout << "CUDA shared filtering execution time: " << filterDuration << " μs" << std::endl;

	t3 = std::chrono::high_resolution_clock::now();

	// Transfer resulting image back
	hipMemcpy(outImage, d_outImagePtr, outImageSize, hipMemcpyDeviceToHost);

	t4 = std::chrono::high_resolution_clock::now();
	copyDuration += std::chrono::duration_cast<std::chrono::microseconds>( t4 - t3 ).count();
	std::cout << "Copy Execution time: " << copyDuration << " μs" << std::endl;

	err = hipGetLastError();
	if (err != hipSuccess) {
		printf("Error: %s\n", hipGetErrorString(err));
		return false;
	}

	// Cleanup after kernel execution
	hipFree(d_sourceImagePtr);
	hipFree(d_outImagePtr);

	return true;
}
